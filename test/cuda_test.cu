#include <iostream>
#include <hip/hip_runtime.h>

// daxpy kernel: y[i] = a * x[i] + y[i]
__global__ void daxpy(int n, double a, const double* x, double* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main()
{
    // Report device count
    int deviceCount = 0;
    hipError_t status = hipGetDeviceCount(&deviceCount);
    if (status != hipSuccess) {
        std::cout << "hipGetDeviceCount error: "
                  << hipGetErrorString(status) << std::endl;
        return 1;
    }
    std::cout << "Detected " << deviceCount 
              << " CUDA device(s)" << std::endl;
    if (deviceCount == 0) {
        return 0;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device 0: " << prop.name << std::endl;

    // Parameters for daxpy
    const int N = 1 << 20;     // 1 million elements
    const double a = 2.5;

    // Host allocations
    double *h_x = new double[N];
    double *h_y = new double[N];
    for (int i = 0; i < N; ++i) {
        h_x[i] = 1.0;
        h_y[i] = 2.0;
    }

    // Device allocations
    double *d_x = nullptr, *d_y = nullptr;
    hipMalloc(&d_x, N * sizeof(double));
    hipMalloc(&d_y, N * sizeof(double));

    // Copy data to device
    hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel: use 256 threads per block
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    daxpy<<<blocks, threadsPerBlock>>>(N, a, d_x, d_y);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy result back and check
    hipMemcpy(h_y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "y[0] = " << h_y[0]
              << "  (expected " << (a * 1.0 + 2.0) << ")" << std::endl;

    // Clean up
    delete[] h_x;
    delete[] h_y;
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
